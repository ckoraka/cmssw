#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include "RecoEgamma/EgammaElectronProducers/interface/TestCUDA.h"

namespace EgammaCUDA {

    __global__ void hello_world_gpu() {
        printf("Hello World from the GPU ");
        if ( blockIdx.x < 100 && threadIdx.x < 100 ) 
            printf("Hello World from the GPU at block %u, thread %u \n", blockIdx.x, threadIdx.x);
    }

    __global__ void printEcalSCkernel(unsigned int nSCs, EcalSC::EcalSCSoA* SCs) {
        size_t firstElement = threadIdx.x; 
        for (unsigned int isc = firstElement; isc < nSCs ; isc += blockDim.x){
            printf("Hello World from the GPU at block %u, thread %u \n", blockIdx.x, threadIdx.x);
            printf("Block dimention is %u \n",blockDim.x );
            printf("superClusRef->seed()->position().theta() : %lf \n",SCs->scTheta(isc) );
        }
        __syncthreads();
    }


    #ifdef __HIPCC__
        void hello_world_gpu_Wrapper() 
        {
            /* Call GPU function */
            const int n_blocks  = 1;
            const int n_threads = 32;
            dim3 grid_dim(n_blocks);
            dim3 block_dim(n_threads);
            std::cout<<" I am in ElectronNHitSeedProducerCUDA::produce "<<std::endl;
            hello_world_gpu<<<grid_dim, block_dim>>>();
            hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if (error != hipSuccess) {
                fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
            }
        }

        void ecalScWrapper(unsigned int nSCs, EcalSC::EcalSCSoA* SCs, hipStream_t stream)
        {
            unsigned int blockSize = 32; //Must be less that 1024 for the T4
            unsigned int gridSize  = (nSCs*blockSize-1)/blockSize; //A good practice is size*blockSize-1 / blockSize 
            printf("Number of SCs : %u \n", nSCs);
            printEcalSCkernel<<<gridSize, blockSize,0,stream>>>(nSCs, SCs);
            hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if (error != hipSuccess) {
                fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
            }

            std::cout<<" Exit the device "<<std::endl;
        }

    #endif
} // End of namespace