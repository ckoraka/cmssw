#include "hip/hip_runtime.h"
#include "RecoEgamma/EgammaElectronAlgos/interface/TrajSeedMatcherCUDA.h"

//CUDA dependencies and libaries
#include <hip/hip_runtime.h>
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"

//C++ libaries
#include <stdio.h>
#include <vector>
#include <cmath>

namespace TrajSeedMatcherCUDA{

//Kernel(s)
  __global__ void match(float *binArr, float &cutVal, const float etaVal, int binSize){
 

  }
  
  
  #ifdef __HIPCC__
    void wrapper(){}     
  #endif
  
}
